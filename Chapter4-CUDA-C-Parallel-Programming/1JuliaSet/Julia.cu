#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>


// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#include "../common/cpu_bitmap.h"
#include "../common/book.h"

#define DIM 1000

struct hipComplex {
  float r;
  float i;
  __device__ hipComplex( float a, float b) : r(a), i(b) {}

  __device__ float magnitude2( void ) {
    return r * r + i * i;
  }
  __device__ hipComplex operator*(const hipComplex& a) {
    return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
  }
  __device__ hipComplex operator+(const hipComplex& a) {
    return hipComplex(r+a.r, i+a.i);
  }
};

__device__ int julia( int x, int y) {
  const float scale = 1.5;
  float jx = scale * (float)(DIM/2 - x)/(DIM/2);
  float jy = scale * (float)(DIM/2 - y)/(DIM/2);

  hipComplex c(-0.8,0.154);
  hipComplex a(jx,jy);

  int i = 0;
  for (i=0; i<200; i++) {
    a = a*a + c;
    if (a.magnitude2() > 1000)
      return 0; // return 0 if it is not in set
  }
  return 1; // return 1 if point is in set
}


__global__ void kernel( unsigned char *ptr) {
  // map from threadIdx/BlockIdx to pixel position
  int x = blockIdx.x;
  int y = blockIdx.y;
  int offset = x+y*gridDim.x;

  // now calculate the value at that position
  int juliaValue = julia(x,y);
  ptr[offset*4 + 0] = 255 * juliaValue;  // red if julia() returns 1, black if pt. not in set
  ptr[offset*4 + 1] = 0;
  ptr[offset*4 + 2] = 0;
  ptr[offset*4 + 3] = 255;
}

int main(void) {
  CPUBitmap bitmap( DIM, DIM );
  unsigned char *dev_bitmap;

  checkCudaErrors(hipMalloc((void**)&dev_bitmap, bitmap.image_size()));

  dim3 grid(DIM,DIM);

  kernel<<<grid,1>>>(dev_bitmap);

  checkCudaErrors(hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost ));
  
  bitmap.display_and_exit();
  printf("size %ld is done\n", bitmap.image_size());
  checkCudaErrors( hipFree(dev_bitmap));
}